#include "hip/hip_runtime.h"
// **************************************************************************
//                                   sw.cu
//                             -------------------
//                           W. Michael Brown (ORNL)
//
//  Device code for acceleration of the sw pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : Tue March 26, 2013
//    email                : brownw@ornl.gov
// ***************************************************************************/

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

#define THIRD (numtyp)0.66666667

#if (ARCH < 300)

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    __local acctyp red_acc[6][BLOCK_ELLIPSE];                               \
    red_acc[0][tid]=f.x;                                                    \
    red_acc[1][tid]=f.y;                                                    \
    red_acc[2][tid]=f.z;                                                    \
    red_acc[3][tid]=energy;                                                 \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      if (offset < s) {                                                     \
        for (int r=0; r<4; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    f.x=red_acc[0][tid];                                                    \
    f.y=red_acc[1][tid];                                                    \
    f.z=red_acc[2][tid];                                                    \
    energy=red_acc[3][tid];                                                 \
    if (vflag>0) {                                                          \
      for (int r=0; r<6; r++)                                               \
        red_acc[r][tid]=virial[r];                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
        if (offset < s) {                                                   \
          for (int r=0; r<6; r++)                                           \
            red_acc[r][tid] += red_acc[r][tid+s];                           \
        }                                                                   \
      }                                                                     \
      for (int r=0; r<6; r++)                                               \
        virial[r]=red_acc[r][tid];                                          \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    engv+=ii;                                                               \
    if (eflag>0) {                                                          \
      *engv+=energy*(acctyp)0.5;                                            \
      engv+=inum;                                                           \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *engv+=virial[i]*(acctyp)0.5;                                       \
        engv+=inum;                                                         \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#else

#define store_answers_p(f, energy, virial, ii, inum, tid, t_per_atom, offset, \
                      eflag, vflag, ans, engv)                              \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
        f.x += shfl_xor(f.x, s, t_per_atom);                                \
        f.y += shfl_xor(f.y, s, t_per_atom);                                \
        f.z += shfl_xor(f.z, s, t_per_atom);                                \
        energy += shfl_xor(energy, s, t_per_atom);                          \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                       \
          for (int r=0; r<6; r++)                                           \
            virial[r] += shfl_xor(virial[r], s, t_per_atom);                \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0) {                                                          \
    engv+=ii;                                                               \
    if (eflag>0) {                                                          \
      *engv+=energy*(acctyp)0.5;                                            \
      engv+=inum;                                                           \
    }                                                                       \
    if (vflag>0) {                                                          \
      for (int i=0; i<6; i++) {                                             \
        *engv+=virial[i]*(acctyp)0.5;                                       \
        engv+=inum;                                                         \
      }                                                                     \
    }                                                                       \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#endif


__kernel void k_sw(const __global numtyp4 *restrict x_, 
                   const __global int * dev_nbor, 
                   const __global int * dev_packed, 
                   __global acctyp4 *restrict ans, 
                   __global acctyp *restrict engv, 
                   const int eflag, const int vflag, const int inum, 
                   const int nbor_pitch, const int t_per_atom, 
                   const numtyp sw_cut, const numtyp sw_epsilon, 
                   const numtyp sw_sigma, const numtyp sw_biga, 
                   const numtyp sw_bigb, const numtyp sw_powerp, 
                   const numtyp sw_powerq, const numtyp sw_cutsq) {

  __local int n_stride;
  __local numtyp pre_sw_c1, pre_sw_c2, pre_sw_c3, pre_sw_c4;
  __local numtyp pre_sw_c5, pre_sw_c6;
  pre_sw_c1=sw_biga*sw_epsilon*sw_powerp*sw_bigb*
            pow(sw_sigma,sw_powerp);
  pre_sw_c2=sw_biga*sw_epsilon*sw_powerq*
            pow(sw_sigma,sw_powerq);
  pre_sw_c3=sw_biga*sw_epsilon*sw_bigb*
            pow(sw_sigma,sw_powerp+(numtyp)1.0);
  pre_sw_c4=sw_biga*sw_epsilon*
            pow(sw_sigma,sw_powerq+(numtyp)1.0);
  pre_sw_c5=sw_biga*sw_epsilon*sw_bigb*
            pow(sw_sigma,sw_powerp);
  pre_sw_c6=sw_biga*sw_epsilon*
            pow(sw_sigma,sw_powerq);

  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  
  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();
  
  if (ii<inum) {
    const __global int *nbor, *list_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,list_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    //int iw=ix.w;
    //int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor<list_end; nbor+=n_stride) {
  
      int j=*nbor;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      //int mtype=itype+jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
        
      if (rsq<sw_cutsq) {
        numtyp r=ucl_sqrt(rsq);
        numtyp rp=ucl_powr(r,-sw_powerp);
        numtyp rq=ucl_powr(r,-sw_powerq);
        numtyp rainv=ucl_recip(r-sw_cut);
        numtyp expsrainv=ucl_exp(sw_sigma*rainv);
        rainv*=rainv*r;
        numtyp force = (pre_sw_c1*rp-pre_sw_c2*rq +
                       (pre_sw_c3*rp-pre_sw_c4*rq) * rainv)*
                       expsrainv*ucl_recip(rsq);
      
        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) 
          energy+=(pre_sw_c5*rp - pre_sw_c6*rq) * expsrainv; 

        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

#define threebody(delr1x, delr1y, delr1z, eflag, energy)                     \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - sw_cut);                                \
  numtyp gsrainv1 = sw_sigma_gamma * rainv1;                             \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rinvsq2 = ucl_recip(rsq2);                                          \
  numtyp rainv2 = ucl_recip(r2 - sw_cut);                                \
  numtyp gsrainv2 = sw_sigma_gamma * rainv2;                             \
  numtyp gsrainvsq2 = gsrainv2*rainv2/r2;                                    \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - sw_costheta;                                       \
  numtyp delcssq = delcs*delcs;                                              \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = sw_lambda_epsilon * facexp*delcssq;                    \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp frad2 = facrad*gsrainvsq2;                                          \
  numtyp facang = sw_lambda_epsilon2 * facexp*delcs;                     \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
                                                                             \
  numtyp csfac2 = rinvsq2*csfacang;                                          \
                                                                             \
  fkx = delr2x*(frad2+csfac2)-delr1x*facang12;                               \
  fky = delr2y*(frad2+csfac2)-delr1y*facang12;                               \
  fkz = delr2z*(frad2+csfac2)-delr1z*facang12;                               \
                                                                             \
  if (eflag>0)                                                               \
    energy+=facrad;                                                          \
  if (vflag>0) {                                                             \
    virial[0] += delr1x*fjx + delr2x*fkx;                                    \
    virial[1] += delr1y*fjy + delr2y*fky;                                    \
    virial[2] += delr1z*fjz + delr2z*fkz;                                    \
    virial[3] += delr1x*fjy + delr2x*fky;                                    \
    virial[4] += delr1x*fjz + delr2x*fkz;                                    \
    virial[5] += delr1y*fjz + delr2y*fkz;                                    \
  }                                                                          \
}

#define threebody_half(delr1x, delr1y, delr1z)                               \
{                                                                            \
  numtyp r1 = ucl_sqrt(rsq1);                                                \
  numtyp rinvsq1 = ucl_recip(rsq1);                                          \
  numtyp rainv1 = ucl_recip(r1 - sw_cut);                                \
  numtyp gsrainv1 = sw_sigma_gamma * rainv1;                             \
  numtyp gsrainvsq1 = gsrainv1*rainv1/r1;                                    \
  numtyp expgsrainv1 = ucl_exp(gsrainv1);                                    \
                                                                             \
  numtyp r2 = ucl_sqrt(rsq2);                                                \
  numtyp rainv2 = ucl_recip(r2 - sw_cut);                                \
  numtyp gsrainv2 = sw_sigma_gamma * rainv2;                             \
  numtyp expgsrainv2 = ucl_exp(gsrainv2);                                    \
                                                                             \
  numtyp rinv12 = ucl_recip(r1*r2);                                          \
  numtyp cs = (delr1x*delr2x + delr1y*delr2y + delr1z*delr2z) * rinv12;      \
  numtyp delcs = cs - sw_costheta;                                       \
  numtyp delcssq = delcs*delcs;                                              \
                                                                             \
  numtyp facexp = expgsrainv1*expgsrainv2;                                   \
                                                                             \
  numtyp facrad = sw_lambda_epsilon * facexp*delcssq;                    \
  numtyp frad1 = facrad*gsrainvsq1;                                          \
  numtyp facang = sw_lambda_epsilon2 * facexp*delcs;                     \
  numtyp facang12 = rinv12*facang;                                           \
  numtyp csfacang = cs*facang;                                               \
  numtyp csfac1 = rinvsq1*csfacang;                                          \
                                                                             \
  fjx = delr1x*(frad1+csfac1)-delr2x*facang12;                               \
  fjy = delr1y*(frad1+csfac1)-delr2y*facang12;                               \
  fjz = delr1z*(frad1+csfac1)-delr2z*facang12;                               \
}

__kernel void k_sw_three_center(const __global numtyp4 *restrict x_, 
                                const __global int * dev_nbor, 
                                const __global int * dev_packed, 
                                __global acctyp4 *restrict ans, 
                                __global acctyp *restrict engv, 
                                const int eflag, const int vflag, 
                                const int inum,  const int nbor_pitch, 
                                const int t_per_atom, const int evatom,
                                const numtyp sw_cut, const numtyp sw_epsilon,
                                const numtyp sw_sigma, const numtyp sw_lambda,
                                const numtyp sw_gamma, const numtyp sw_costheta,
                                const numtyp sw_cutsq) {
  __local int tpa_sq, n_stride;
  __local numtyp sw_sigma_gamma, sw_lambda_epsilon;
  __local numtyp sw_lambda_epsilon2;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  sw_sigma_gamma=sw_sigma*sw_gamma;
  sw_lambda_epsilon=sw_lambda*sw_epsilon;
  sw_lambda_epsilon2=(numtyp)2.0*sw_lambda_epsilon;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);
  
  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();
  
  if (ii<inum) {
    const __global int *nbor_j, *list_end;
    int i, numj;

    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,list_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    //int iw=ix.w;
    //int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor_j<list_end; nbor_j+=n_stride) {
  
      int j=*nbor_j;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      //int mtype=itype+jx.w;

      // Compute r12
      numtyp delr1x = jx.x-ix.x;
      numtyp delr1y = jx.y-ix.y;
      numtyp delr1z = jx.z-ix.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;
        
      if (rsq1 > sw_cutsq) continue;

      const __global int *nbor_k=nbor_j-offset_j+offset_k;
      if (nbor_k<=nbor_j)
        nbor_k+=n_stride;

      for ( ; nbor_k<list_end; nbor_k+=n_stride) {
        int k=*nbor_k;
        k &= NEIGHMASK;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        numtyp delr2x = kx.x-ix.x;
        numtyp delr2y = kx.y-ix.y;
        numtyp delr2z = kx.z-ix.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        if (rsq2 < sw_cutsq) {
          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x -= fjx + fkx;
          f.y -= fjy + fky;
          f.z -= fjz + fkz;
        }  
      }
    } // for nbor

    numtyp pre;
    if (evatom==1)
      pre=THIRD;
    else
      pre=(numtyp)2.0;
    energy*=pre;
    for (int i=0; i<6; i++)
      virial[i]*=pre;

    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);

  } // if ii
}

__kernel void k_sw_three_end(const __global numtyp4 *restrict x_, 
                             const __global int * dev_nbor, 
                             const __global int * dev_packed, 
                             __global acctyp4 *restrict ans, 
                             __global acctyp *restrict engv, 
                             const int eflag, const int vflag, 
                             const int inum,  const int nbor_pitch, 
                             const int t_per_atom, const numtyp sw_cut, 
                             const numtyp sw_epsilon, const numtyp sw_sigma,
                             const numtyp sw_lambda, const numtyp sw_gamma,
                             const numtyp sw_costheta, const numtyp sw_cutsq) {
  __local int tpa_sq, n_stride;
  __local numtyp sw_sigma_gamma, sw_lambda_epsilon;
  __local numtyp sw_lambda_epsilon2;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  sw_sigma_gamma=sw_sigma*sw_gamma;
  sw_lambda_epsilon=sw_lambda*sw_epsilon;
  sw_lambda_epsilon2=(numtyp)2.0*sw_lambda_epsilon;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);
  
  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();
  
  if (ii<inum) {
    const __global int *nbor_j, *list_end, *k_end;
    int i, numj;

    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,list_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    //int iw=ix.w;
    //int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor_j<list_end; nbor_j+=n_stride) {
      int j=*nbor_j;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      //int mtype=itype+jx.w;

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;
        
      if (rsq1 > sw_cutsq) continue;

      const __global int *nbor_k=dev_nbor+j+nbor_pitch;
      int numk=*nbor_k; 
      if (dev_nbor==dev_packed) {
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k+=nbor_pitch;
        nbor_k=dev_packed+*nbor_k;
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=*nbor_k;
        k &= NEIGHMASK;

        if (k == i)
          continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        if (rsq2 < sw_cutsq) {
          numtyp fjx, fjy, fjz;
          //if (evatom==0) {
            threebody_half(delr1x,delr1y,delr1z);
          //} else {
          //  numtyp fkx, fky, fkz;
          //  threebody(delr1x,delr1y,delr1z,eflag,energy);
          //}

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

__kernel void k_sw_three_end_vatom(const __global numtyp4 *restrict x_, 
                             const __global int * dev_nbor, 
                             const __global int * dev_packed, 
                             __global acctyp4 *restrict ans, 
                             __global acctyp *restrict engv, 
                             const int eflag, const int vflag, 
                             const int inum,  const int nbor_pitch, 
                             const int t_per_atom, const numtyp sw_cut, 
                             const numtyp sw_epsilon, const numtyp sw_sigma,
                             const numtyp sw_lambda, const numtyp sw_gamma,
                             const numtyp sw_costheta, const numtyp sw_cutsq) {
  __local int tpa_sq, n_stride;
  __local numtyp sw_sigma_gamma, sw_lambda_epsilon;
  __local numtyp sw_lambda_epsilon2;
  tpa_sq=fast_mul(t_per_atom,t_per_atom);
  sw_sigma_gamma=sw_sigma*sw_gamma;
  sw_lambda_epsilon=sw_lambda*sw_epsilon;
  sw_lambda_epsilon2=(numtyp)2.0*sw_lambda_epsilon;

  int tid, ii, offset;
  atom_info(tpa_sq,ii,tid,offset);
  
  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();
  
  if (ii<inum) {
    const __global int *nbor_j, *list_end, *k_end;
    int i, numj;

    int offset_j=offset/t_per_atom;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset_j,i,numj,
              n_stride,list_end,nbor_j);
    int offset_k=tid & (t_per_atom-1);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    //int iw=ix.w;
    //int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    for ( ; nbor_j<list_end; nbor_j+=n_stride) {
      int j=*nbor_j;
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      //int mtype=itype+jx.w;

      // Compute r12
      numtyp delr1x = ix.x-jx.x;
      numtyp delr1y = ix.y-jx.y;
      numtyp delr1z = ix.z-jx.z;
      numtyp rsq1 = delr1x*delr1x+delr1y*delr1y+delr1z*delr1z;
        
      if (rsq1 > sw_cutsq) continue;

      const __global int *nbor_k=dev_nbor+j+nbor_pitch;
      int numk=*nbor_k; 
      if (dev_nbor==dev_packed) {
        nbor_k+=nbor_pitch+fast_mul(j,t_per_atom-1);
        k_end=nbor_k+fast_mul(numk/t_per_atom,n_stride)+(numk & (t_per_atom-1));
        nbor_k+=offset_k;
      } else {
        nbor_k+=nbor_pitch;
        nbor_k=dev_packed+*nbor_k;
        k_end=nbor_k+numk;
        nbor_k+=offset_k;
      }

      for ( ; nbor_k<k_end; nbor_k+=n_stride) {
        int k=*nbor_k;
        k &= NEIGHMASK;

        if (k == i)
          continue;

        numtyp4 kx; fetch4(kx,k,pos_tex);
        numtyp delr2x = kx.x - jx.x;
        numtyp delr2y = kx.y - jx.y;
        numtyp delr2z = kx.z - jx.z;
        numtyp rsq2 = delr2x*delr2x + delr2y*delr2y + delr2z*delr2z;
        if (rsq2 < sw_cutsq) {
          numtyp fjx, fjy, fjz, fkx, fky, fkz;
          threebody(delr1x,delr1y,delr1z,eflag,energy);

          f.x += fjx;
          f.y += fjy;
          f.z += fjz;
        }
      }

    } // for nbor
    energy*=THIRD;
    for (int i=0; i<6; i++)
      virial[i]*=THIRD;
    #ifdef THREE_CONCURRENT
    store_answers(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                  eflag,vflag,ans,engv);
    #else
    store_answers_p(f,energy,virial,ii,inum,tid,tpa_sq,offset,
                    eflag,vflag,ans,engv);
    #endif
  } // if ii
}

