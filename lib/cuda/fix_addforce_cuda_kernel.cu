#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ F_FLOAT sharedmem[];


__global__ void Cuda_FixAddForceCuda_PostForce_Kernel(int groupbit, F_FLOAT xvalue, F_FLOAT yvalue, F_FLOAT zvalue)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 3 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit)
      //if (iregion >= 0 &&
      //match(x[i][0],x[i][1],x[i][2],iregion)) //currently not supported
    {
      sharedmem[threadIdx.x] = -xvalue * _x[i] - yvalue * _x[i + 1 * _nmax] - zvalue * _x[i + 2 * _nmax];
      sharedmem[threadIdx.x + blockDim.x] = _f[i];
      sharedmem[threadIdx.x + 2 * blockDim.x] = _f[i + 1 * _nmax];
      sharedmem[threadIdx.x + 3 * blockDim.x] = _f[i + 2 * _nmax];
      _f[i] += xvalue;
      _f[i + 1 * _nmax] += yvalue;
      _f[i + 2 * _nmax] += zvalue;
    }

  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  reduceBlock(&sharedmem[3 * blockDim.x]);
  F_FLOAT* buffer = (F_FLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = sharedmem[3 * blockDim.x];
  }

}


__global__ void reduce_foriginal(int n, F_FLOAT* foriginal)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  F_FLOAT myforig = 0.0;
  F_FLOAT* buf = (F_FLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    foriginal[blockIdx.x] = myforig;
}
