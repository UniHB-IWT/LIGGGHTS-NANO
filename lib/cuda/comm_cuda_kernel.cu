#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__global__ void Cuda_CommCuda_PackComm_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_FLOAT dx, X_FLOAT dy, X_FLOAT dz, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    ((X_FLOAT*) buffer)[i] = _x[j] + dx;
    ((X_FLOAT*) buffer)[i + 1 * n] = _x[j + _nmax] + dy;
    ((X_FLOAT*) buffer)[i + 2 * n] = _x[j + 2 * _nmax] + dz;
  }
}

__global__ void Cuda_CommCuda_PackCommVel_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_FLOAT dx, X_FLOAT dy, X_FLOAT dz, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    ((X_FLOAT*) buffer)[i] = _x[j] + dx;
    ((X_FLOAT*) buffer)[i + 1 * n] = _x[j + _nmax] + dy;
    ((X_FLOAT*) buffer)[i + 2 * n] = _x[j + 2 * _nmax] + dz;
    ((X_FLOAT*) buffer)[i + 3 * n] = _v[j];
    ((X_FLOAT*) buffer)[i + 4 * n] = _v[j + _nmax];
    ((X_FLOAT*) buffer)[i + 5 * n] = _v[j + 2 * _nmax];
  }
}

__global__ void Cuda_CommCuda_PackComm_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_FLOAT dx, X_FLOAT dy, X_FLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = i;
    j = list[i];

    _x[i + first] = _x[j] + dx;
    _x[i + first + _nmax] = _x[j + _nmax] + dy;
    _x[i + first + 2 * _nmax] = _x[j + 2 * _nmax] + dz;
  }
}

__global__ void Cuda_CommCuda_PackCommVel_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_FLOAT dx, X_FLOAT dy, X_FLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = i;
    j = list[i];

    _x[i + first] = _x[j] + dx;
    _x[i + first + _nmax] = _x[j + _nmax] + dy;
    _x[i + first + 2 * _nmax] = _x[j + 2 * _nmax] + dz;
    _v[i + first] = _v[j];
    _v[i + first + _nmax] = _v[j + _nmax];
    _v[i + first + 2 * _nmax] = _v[j + 2 * _nmax];
  }
}

__global__ void Cuda_CommCuda_UnpackComm_Kernel(int n, int first, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    _x[i + first] = ((X_FLOAT*) buffer)[i];
    _x[i + first + _nmax] = ((X_FLOAT*) buffer)[i + 1 * n];
    _x[i + first + 2 * _nmax] = ((X_FLOAT*) buffer)[i + 2 * n];
  }
}


__global__ void Cuda_CommCuda_UnpackCommVel_Kernel(int n, int first, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    _x[i + first] = ((X_FLOAT*) buffer)[i];
    _x[i + first + _nmax] = ((X_FLOAT*) buffer)[i + 1 * n];
    _x[i + first + 2 * _nmax] = ((X_FLOAT*) buffer)[i + 2 * n];
    _v[i + first] = ((X_FLOAT*) buffer)[i + 3 * n];
    _v[i + first + _nmax] = ((X_FLOAT*) buffer)[i + 4 * n];
    _v[i + first + 2 * _nmax] = ((X_FLOAT*) buffer)[i + 5 * n];
  }
}

__global__ void Cuda_CommCuda_PackReverse_Kernel(int n, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    ((F_FLOAT*) _buffer)[i] = _f[i + first];
    ((F_FLOAT*) _buffer)[i + n] = _f[i + first + _nmax];
    ((F_FLOAT*) _buffer)[i + 2 * n] = _f[i + first + 2 * _nmax];
  }

}

__global__ void Cuda_CommCuda_UnpackReverse_Kernel(int* sendlist, int n, int maxlistlength, int iswap)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];
    _f[j] += ((F_FLOAT*)_buffer)[i];
    _f[j + _nmax] += ((F_FLOAT*) _buffer)[i + n];
    _f[j + 2 * _nmax] += ((F_FLOAT*) _buffer)[i + 2 * n];
  }

}

__global__ void Cuda_CommCuda_UnpackReverse_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    _f[j] += _f[i + first];
    _f[j + _nmax] += _f[i + first + _nmax];
    _f[j + 2 * _nmax] += _f[i + first + 2 * _nmax];
  }

}

extern __shared__ int shared[];

__global__ void Cuda_CommCuda_BuildSendlist_Single(int bordergroup, int ineed, int atom_nfirst,
    int nfirst, int nlast, int dim, int iswap, X_FLOAT* slablo, X_FLOAT* slabhi, int* sendlist, int maxlistlength)
{
  int* list = sendlist + iswap * maxlistlength;
  X_FLOAT lo = slablo[iswap];
  X_FLOAT hi = slabhi[iswap];
  bool add = false;

  if(!bordergroup || ineed >= 2) {
    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x + nfirst;

    if(i < nlast)
      if(_x[i + dim * _nmax] >= lo && _x[i + dim * _nmax] <= hi) {
        add = true;
      }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;


  } else {

    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

    if(i < atom_nfirst)
      if(_x[i + dim * _nmax] >= lo && _x[i + dim * _nmax] <= hi) {
        add = true;
      }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

    __syncthreads();

    add = false;
    i += _nlocal;

    if(i < nlast)
      if(_x[i + dim * _nmax] >= lo && _x[i + dim * _nmax] <= hi) {
        add = true;
      }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

  }
}


__global__ void Cuda_CommCuda_BuildSendlist_Multi(int bordergroup, int ineed, int atom_nfirst
    , int nfirst, int nlast, int dim, int iswap, X_FLOAT* multilo, X_FLOAT* multihi, int* sendlist, int maxlistlength)
{
  int* list = sendlist + iswap * maxlistlength;
  X_FLOAT* mlo = &multilo[iswap * _cuda_ntypes];
  X_FLOAT* mhi = &multihi[iswap * _cuda_ntypes];
  int itype = 0;
  bool add = false;

  if(!bordergroup || ineed >= 2) {
    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x + nfirst;

    if(i < nlast) {
      itype = _type[i];

      if(_x[i + dim * _nmax] >= mlo[itype] && _x[i + dim * _nmax] <= mhi[itype]) {
        add = true;
      }
    }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;


  } else {

    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

    if(i < atom_nfirst) {
      itype = _type[i];

      if(_x[i + dim * _nmax] >= mlo[itype] && _x[i + dim * _nmax] <= mhi[itype]) {
        add = true;
      }
    }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

    __syncthreads();

    add = false;
    i += _nlocal;

    if(i < nlast) {
      itype = _type[i];

      if(_x[i + dim * _nmax] >= mlo[itype] && _x[i + dim * _nmax] <= mhi[itype]) {
        add = true;
      }
    }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

  }
}
