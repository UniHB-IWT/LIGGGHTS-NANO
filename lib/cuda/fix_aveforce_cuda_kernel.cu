#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ F_FLOAT sharedmem[];


__global__ void Cuda_FixAveForceCuda_PostForce_FOrg_Kernel(int groupbit)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 3 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      sharedmem[threadIdx.x] = _f[i];
      sharedmem[threadIdx.x + blockDim.x] = _f[i + 1 * _nmax];
      sharedmem[threadIdx.x + 2 * blockDim.x] = _f[i + 2 * _nmax];
      sharedmem[threadIdx.x + 3 * blockDim.x] = 1;
    }

  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  reduceBlock(&sharedmem[3 * blockDim.x]);
  F_FLOAT* buffer = (F_FLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = sharedmem[3 * blockDim.x];
  }
}


__global__ void Cuda_FixAveForceCuda_reduce_foriginal(int n, F_FLOAT* foriginal)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  F_FLOAT myforig = 0.0;
  F_FLOAT* buf = (F_FLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    foriginal[blockIdx.x] = myforig;
}

__global__ void Cuda_FixAveForceCuda_PostForce_Set_Kernel(int groupbit, int xflag, int yflag, int zflag, F_FLOAT xvalue, F_FLOAT yvalue, F_FLOAT zvalue)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      if(xflag) _f[i] = xvalue;

      if(yflag) _f[i + 1 * _nmax] = yvalue;

      if(zflag) _f[i + 2 * _nmax] = zvalue;
    }
}
