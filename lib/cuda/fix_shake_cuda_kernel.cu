#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__device__ void v_tally(int &vflag_global, int &vflag_atom, int &n, int* list, ENERGY_FLOAT total, ENERGY_FLOAT* v)
{
  /*if(vflag_global)
  {
    ENERGY_FLOAT fraction = n/total;
  ENERGY_FLOAT* shared = &sharedmem[threadIdx.x];
    *shared += fraction*v[0]; shared+=blockDim.x;
    *shared += fraction*v[1]; shared+=blockDim.x;
    *shared += fraction*v[2]; shared+=blockDim.x;
    *shared += fraction*v[3]; shared+=blockDim.x;
    *shared += fraction*v[4]; shared+=blockDim.x;
    *shared += fraction*v[5];
  }*/
  if(vflag_atom) {
    ENERGY_FLOAT fraction = ENERGY_F(1.0) / total;

    for(int i = 0; i < n; i++) {
      int m = list[i];
      ENERGY_FLOAT* myvatom = &_vatom[m];

      *myvatom += fraction * v[0];
      myvatom += _nmax;
      *myvatom += fraction * v[1];
      myvatom += _nmax;
      *myvatom += fraction * v[2];
      myvatom += _nmax;
      *myvatom += fraction * v[3];
      myvatom += _nmax;
      *myvatom += fraction * v[4];
      myvatom += _nmax;
      *myvatom += fraction * v[5];
    }
  }
}

inline __device__ void minimum_image(X_FLOAT3 &delta)
{
  if(_triclinic == 0) {
    if(_periodicity[0]) {
      delta.x += delta.x < -X_F(0.5) * _prd[0] ? _prd[0] :
                 (delta.x >  X_F(0.5) * _prd[0] ? -_prd[0] : X_F(0.0));
    }

    if(_periodicity[1]) {
      delta.y += delta.y < -X_F(0.5) * _prd[1] ? _prd[1] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_prd[1] : X_F(0.0));
    }

    if(_periodicity[2]) {
      delta.z += delta.z < -X_F(0.5) * _prd[2] ? _prd[2] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_prd[2] : X_F(0.0));
    }

  } else {
    if(_periodicity[1]) {
      delta.z += delta.z < -X_F(0.5) * _prd[2] ? _prd[2] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_prd[2] : X_F(0.0));
      delta.y += delta.z < -X_F(0.5) * _prd[2] ? _h[3] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_h[3] : X_F(0.0));
      delta.x += delta.z < -X_F(0.5) * _prd[2] ? _h[4] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_h[4] : X_F(0.0));

    }

    if(_periodicity[1]) {
      delta.y += delta.y < -X_F(0.5) * _prd[1] ? _prd[1] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_prd[1] : X_F(0.0));
      delta.x += delta.y < -X_F(0.5) * _prd[1] ? _h[5] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_h[5] : X_F(0.0));

    }

    if(_periodicity[0]) {
      delta.x += delta.x < -X_F(0.5) * _prd[0] ? _prd[0] :
                 (delta.x >  X_F(0.5) * _prd[0] ? -_prd[0] : X_F(0.0));
    }
  }
}

__global__ void FixShakeCuda_UnconstrainedUpdate_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i >= _nlocal) return;

  X_FLOAT3 my_xshake = {X_F(0.0), X_F(0.0), X_F(0.0)};

  if(_shake_flag[i]) {
    F_FLOAT* my_f = _f + i;
    V_FLOAT* my_v = _v + i;
    X_FLOAT* my_x = _x + i;

    V_FLOAT 		dtfmsq = _dtfsq;

    if(_rmass_flag) dtfmsq *= V_F(1.0) / _rmass[i];
    else 			dtfmsq *= V_F(1.0) / _mass[_type[i]];

    my_xshake.x =  *my_x + _dtv* *my_v + dtfmsq* *my_f;
    my_f += _nmax;
    my_v += _nmax;
    my_x += _nmax;
    my_xshake.y =  *my_x + _dtv* *my_v + dtfmsq* *my_f;
    my_f += _nmax;
    my_v += _nmax;
    my_x += _nmax;
    my_xshake.z =  *my_x + _dtv* *my_v + dtfmsq* *my_f;
  }

  _xshake[i] = my_xshake;
}




__device__ void FixShakeCuda_Shake2(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[2];
  ENERGY_FLOAT v[6];
  X_FLOAT invmass0, invmass1;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  X_FLOAT bond1 = _bond_distance[_shake_type[m]];

  // r01 = distance vec between atoms, with PBC

  X_FLOAT3 r01;

  X_FLOAT4 x_i0, x_i1;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  // s01 = distance vec after unconstrained update, with PBC

  X_FLOAT3 s01;
  X_FLOAT3 xs_i0 = _xshake[i0];
  X_FLOAT3 xs_i1 = _xshake[i1];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  // scalar distances between atoms

  X_FLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_FLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
  }

  X_FLOAT a = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_FLOAT b = X_F(2.0) * (invmass0 + invmass1) *
              (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_FLOAT c = s01sq - bond1 * bond1;

  // error check

  X_FLOAT determ = b * b - X_F(4.0) * a * c;

  if(determ < X_F(0.0)) {
    _flag[0]++;
    determ = X_F(0.0);
  }

  // exact quadratic solution for lamda

  X_FLOAT lamda, lamda1, lamda2;
  lamda1 = -b + _SQRT_(determ);
  lamda2 = -lamda1 - X_F(2.0) * b;
  lamda1 *= X_F(1.0) / (X_F(2.0) * a);
  lamda2 *= X_F(1.0) / (X_F(2.0) * a);

  lamda = (fabs(lamda1) <= fabs(lamda2)) ? lamda1 : lamda2;

  // update forces if atom is owned by this processor

  lamda *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0]         += lamda * r01.x;
    _f[i0 + _nmax]   += lamda * r01.y;
    _f[i0 + 2 * _nmax] += lamda * r01.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1]         -= lamda * r01.x;
    _f[i1 + _nmax]   -= lamda * r01.y;
    _f[i1 + 2 * _nmax] -= lamda * r01.z;
    list[nlist++] = i1;
  }

  if(vflag || vflag_atom) {
    ENERGY_FLOAT* shared = &sharedmem[threadIdx.x];
    X_FLOAT factor = nlist;
    v[0] = lamda * r01.x * r01.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda * r01.y * r01.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda * r01.z * r01.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda * r01.x * r01.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda * r01.x * r01.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda * r01.y * r01.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 2.0, v);
  }
}


__device__ void FixShakeCuda_Shake3(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[3];
  ENERGY_FLOAT v[6];
  X_FLOAT invmass0, invmass1, invmass2;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  int i2 = _map_array[_shake_atom[m + 2 * _nmax]];
  X_FLOAT bond1 = _bond_distance[_shake_type[m]];
  X_FLOAT bond2 = _bond_distance[_shake_type[m + _nmax]];

  // r01 = distance vec between atoms, with PBC

  X_FLOAT3 r01, r02;

  X_FLOAT4 x_i0, x_i1, x_i2;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);
  x_i2 = fetchXType(i2);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  r02.x = x_i0.x - x_i2.x;
  r02.y = x_i0.y - x_i2.y;
  r02.z = x_i0.z - x_i2.z;
  minimum_image(r02);

  // s01 = distance vec after unconstrained update, with PBC

  X_FLOAT3 s01, s02;
  X_FLOAT3 xs_i0 = _xshake[i0];
  X_FLOAT3 xs_i1 = _xshake[i1];
  X_FLOAT3 xs_i2 = _xshake[i2];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  s02.x = xs_i0.x - xs_i2.x;
  s02.y = xs_i0.y - xs_i2.y;
  s02.z = xs_i0.z - xs_i2.z;
  minimum_image(s02);

  // scalar distances between atoms

  X_FLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_FLOAT r02sq = r02.x * r02.x + r02.y * r02.y + r02.z * r02.z;
  X_FLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;
  X_FLOAT s02sq = s02.x * s02.x + s02.y * s02.y + s02.z * s02.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
    invmass2 = X_F(1.0) / _rmass[i2];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
    invmass2 = X_F(1.0) / _mass[static_cast <int>(x_i2.w)];
  }

  X_FLOAT a11 = X_F(2.0) * (invmass0 + invmass1) *
                (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_FLOAT a12 = X_F(2.0) * invmass0 *
                (s01.x * r02.x + s01.y * r02.y + s01.z * r02.z);
  X_FLOAT a21 = X_F(2.0) * invmass0 *
                (s02.x * r01.x + s02.y * r01.y + s02.z * r01.z);
  X_FLOAT a22 = X_F(2.0) * (invmass0 + invmass2) *
                (s02.x * r02.x + s02.y * r02.y + s02.z * r02.z);

  // error check

  X_FLOAT determ = a11 * a22 - a12 * a21;

  if(determ == X_F(0.0)) _flag[0]++;

  X_FLOAT determinv = X_F(1.0) / determ;

  X_FLOAT a11inv = a22 * determinv;
  X_FLOAT a12inv = -a12 * determinv;
  X_FLOAT a21inv = -a21 * determinv;
  X_FLOAT a22inv = a11 * determinv;

  // quadratic correction coeffs

  X_FLOAT r0102 = (r01.x * r02.x + r01.y * r02.y + r01.z * r02.z);

  X_FLOAT quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_FLOAT quad1_0202 = invmass0 * invmass0 * r02sq;
  X_FLOAT quad1_0102 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0102;

  X_FLOAT quad2_0202 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
  X_FLOAT quad2_0101 = invmass0 * invmass0 * r01sq;
  X_FLOAT quad2_0102 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0102;

  // iterate until converged

  X_FLOAT lamda01 = X_F(0.0);
  X_FLOAT lamda02 = X_F(0.0);
  int niter = 0;
  int done = 0;

  X_FLOAT quad1, quad2, b1, b2, lamda01_new, lamda02_new;

  //maybe all running full loop?
  while(__any(!done) && niter < _max_iter) {
    quad1 = quad1_0101 * lamda01 * lamda01 + quad1_0202 * lamda02 * lamda02 +
            quad1_0102 * lamda01 * lamda02;
    quad2 = quad2_0101 * lamda01 * lamda01 + quad2_0202 * lamda02 * lamda02 +
            quad2_0102 * lamda01 * lamda02;

    b1 = bond1 * bond1 - s01sq - quad1;
    b2 = bond2 * bond2 - s02sq - quad2;

    lamda01_new = a11inv * b1 + a12inv * b2;
    lamda02_new = a21inv * b1 + a22inv * b2;

    done++;
    done = (fabs(lamda01_new - lamda01) > _tolerance) ? 0 : done;
    done = (fabs(lamda02_new - lamda02) > _tolerance) ? 0 : done;


    lamda01 = done < 2 ? lamda01_new : lamda01;
    lamda02 = done < 2 ? lamda02_new : lamda02;
    niter++;
  }

  // update forces if atom is owned by this processor

  lamda01 *= X_F(1.0) / _dtfsq;
  lamda02 *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0] += lamda01 * r01.x + lamda02 * r02.x;
    _f[i0 + _nmax] += lamda01 * r01.y + lamda02 * r02.y;
    _f[i0 + 2 * _nmax] += lamda01 * r01.z + lamda02 * r02.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1] -= lamda01 * r01.x;
    _f[i1 + _nmax] -= lamda01 * r01.y;
    _f[i1 + 2 * _nmax] -= lamda01 * r01.z;
    list[nlist++] = i1;
  }

  if(i2 < _nlocal) {
    _f[i2] -= lamda02 * r02.x;
    _f[i2 + _nmax] -= lamda02 * r02.y;
    _f[i2 + 2 * _nmax] -= lamda02 * r02.z;
    list[nlist++] = i2;
  }

  if(vflag || vflag_atom) {
    ENERGY_FLOAT* shared = &sharedmem[threadIdx.x];
    X_FLOAT factor = X_F(2.0) / X_F(3.0) * nlist;
    v[0] = lamda01 * r01.x * r01.x + lamda02 * r02.x * r02.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda01 * r01.y * r01.y + lamda02 * r02.y * r02.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda01 * r01.z * r01.z + lamda02 * r02.z * r02.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda01 * r01.x * r01.y + lamda02 * r02.x * r02.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda01 * r01.x * r01.z + lamda02 * r02.x * r02.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda01 * r01.y * r01.z + lamda02 * r02.y * r02.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 3.0, v);
  }
}

__device__ void FixShakeCuda_Shake4(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[4];
  ENERGY_FLOAT v[6];
  X_FLOAT invmass0, invmass1, invmass2, invmass3;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  int i2 = _map_array[_shake_atom[m + 2 * _nmax]];
  int i3 = _map_array[_shake_atom[m + 3 * _nmax]];
  X_FLOAT bond1 = _bond_distance[_shake_type[m]];
  X_FLOAT bond2 = _bond_distance[_shake_type[m + _nmax]];
  X_FLOAT bond3 = _bond_distance[_shake_type[m + 2 * _nmax]];

  // r01 = distance vec between atoms, with PBC

  X_FLOAT3 r01, r02, r03;

  X_FLOAT4 x_i0, x_i1, x_i2, x_i3;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);
  x_i2 = fetchXType(i2);
  x_i3 = fetchXType(i3);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  r02.x = x_i0.x - x_i2.x;
  r02.y = x_i0.y - x_i2.y;
  r02.z = x_i0.z - x_i2.z;
  minimum_image(r02);

  r03.x = x_i0.x - x_i3.x;
  r03.y = x_i0.y - x_i3.y;
  r03.z = x_i0.z - x_i3.z;
  minimum_image(r03);

  // s01 = distance vec after unconstrained update, with PBC

  X_FLOAT3 s01, s02, s03;
  X_FLOAT3 xs_i0 = _xshake[i0];
  X_FLOAT3 xs_i1 = _xshake[i1];
  X_FLOAT3 xs_i2 = _xshake[i2];
  X_FLOAT3 xs_i3 = _xshake[i3];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  s02.x = xs_i0.x - xs_i2.x;
  s02.y = xs_i0.y - xs_i2.y;
  s02.z = xs_i0.z - xs_i2.z;
  minimum_image(s02);

  s03.x = xs_i0.x - xs_i3.x;
  s03.y = xs_i0.y - xs_i3.y;
  s03.z = xs_i0.z - xs_i3.z;
  minimum_image(s03);

  // scalar distances between atoms

  X_FLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_FLOAT r02sq = r02.x * r02.x + r02.y * r02.y + r02.z * r02.z;
  X_FLOAT r03sq = r03.x * r03.x + r03.y * r03.y + r03.z * r03.z;
  X_FLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;
  X_FLOAT s02sq = s02.x * s02.x + s02.y * s02.y + s02.z * s02.z;
  X_FLOAT s03sq = s03.x * s03.x + s03.y * s03.y + s03.z * s03.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
    invmass2 = X_F(1.0) / _rmass[i2];
    invmass3 = X_F(1.0) / _rmass[i3];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
    invmass2 = X_F(1.0) / _mass[static_cast <int>(x_i2.w)];
    invmass3 = X_F(1.0) / _mass[static_cast <int>(x_i3.w)];
  }

  X_FLOAT a11 = X_F(2.0) * (invmass0 + invmass1) *
                (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_FLOAT a12 = X_F(2.0) * invmass0 *
                (s01.x * r02.x + s01.y * r02.y + s01.z * r02.z);
  X_FLOAT a13 = X_F(2.0) * invmass0 *
                (s01.x * r03.x + s01.y * r03.y + s01.z * r03.z);
  X_FLOAT a21 = X_F(2.0) * invmass0 *
                (s02.x * r01.x + s02.y * r01.y + s02.z * r01.z);
  X_FLOAT a22 = X_F(2.0) * (invmass0 + invmass2) *
                (s02.x * r02.x + s02.y * r02.y + s02.z * r02.z);
  X_FLOAT a23 = X_F(2.0) * (invmass0) *
                (s02.x * r03.x + s02.y * r03.y + s02.z * r03.z);
  X_FLOAT a31 = X_F(2.0) * (invmass0) *
                (s03.x * r01.x + s03.y * r01.y + s03.z * r01.z);
  X_FLOAT a32 = X_F(2.0) * (invmass0) *
                (s03.x * r02.x + s03.y * r02.y + s03.z * r02.z);
  X_FLOAT a33 = X_F(2.0) * (invmass0 + invmass3) *
                (s03.x * r03.x + s03.y * r03.y + s03.z * r03.z);

  // error check

  X_FLOAT determ = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 -
                   a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

  if(determ == X_F(0.0)) _flag[0]++;

  X_FLOAT determinv = X_F(1.0) / determ;

  X_FLOAT a11inv = determinv * (a22 * a33 - a23 * a32);
  X_FLOAT a12inv = -determinv * (a12 * a33 - a13 * a32);
  X_FLOAT a13inv = determinv * (a12 * a23 - a13 * a22);
  X_FLOAT a21inv = -determinv * (a21 * a33 - a23 * a31);
  X_FLOAT a22inv = determinv * (a11 * a33 - a13 * a31);
  X_FLOAT a23inv = -determinv * (a11 * a23 - a13 * a21);
  X_FLOAT a31inv = determinv * (a21 * a32 - a22 * a31);
  X_FLOAT a32inv = -determinv * (a11 * a32 - a12 * a31);
  X_FLOAT a33inv = determinv * (a11 * a22 - a12 * a21);

  // quadratic correction coeffs

  X_FLOAT r0102 = (r01.x * r02.x + r01.y * r02.y + r01.z * r02.z);
  X_FLOAT r0103 = (r01.x * r03.x + r01.y * r03.y + r01.z * r03.z);
  X_FLOAT r0203 = (r02.x * r03.x + r02.y * r03.y + r02.z * r03.z);

  X_FLOAT quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_FLOAT quad1_0202 = invmass0 * invmass0 * r02sq;
  X_FLOAT quad1_0303 = invmass0 * invmass0 * r03sq;
  X_FLOAT quad1_0102 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0102;
  X_FLOAT quad1_0103 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0103;
  X_FLOAT quad1_0203 = X_F(2.0) * invmass0 * invmass0 * r0203;

  X_FLOAT quad2_0101 = invmass0 * invmass0 * r01sq;
  X_FLOAT quad2_0202 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
  X_FLOAT quad2_0303 = invmass0 * invmass0 * r03sq;
  X_FLOAT quad2_0102 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0102;
  X_FLOAT quad2_0103 = X_F(2.0) * invmass0 * invmass0 * r0103;
  X_FLOAT quad2_0203 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0203;

  X_FLOAT quad3_0101 = invmass0 * invmass0 * r01sq;
  X_FLOAT quad3_0202 = invmass0 * invmass0 * r02sq;
  X_FLOAT quad3_0303 = (invmass0 + invmass3) * (invmass0 + invmass3) * r03sq;
  X_FLOAT quad3_0102 = X_F(2.0) * invmass0 * invmass0 * r0102;
  X_FLOAT quad3_0103 = X_F(2.0) * (invmass0 + invmass3) * invmass0 * r0103;
  X_FLOAT quad3_0203 = X_F(2.0) * (invmass0 + invmass3) * invmass0 * r0203;
  // iterate until converged

  X_FLOAT lamda01 = X_F(0.0);
  X_FLOAT lamda02 = X_F(0.0);
  X_FLOAT lamda03 = X_F(0.0);
  int niter = 0;
  int done = 0;

  X_FLOAT quad1, quad2, quad3, b1, b2, b3, lamda01_new, lamda02_new, lamda03_new;

  //maybe all running full loop?
  while(__any(!done) && niter < _max_iter) {
    quad1 = quad1_0101 * lamda01 * lamda01 +
            quad1_0202 * lamda02 * lamda02 +
            quad1_0303 * lamda03 * lamda03 +
            quad1_0102 * lamda01 * lamda02 +
            quad1_0103 * lamda01 * lamda03 +
            quad1_0203 * lamda02 * lamda03;

    quad2 = quad2_0101 * lamda01 * lamda01 +
            quad2_0202 * lamda02 * lamda02 +
            quad2_0303 * lamda03 * lamda03 +
            quad2_0102 * lamda01 * lamda02 +
            quad2_0103 * lamda01 * lamda03 +
            quad2_0203 * lamda02 * lamda03;

    quad3 = quad3_0101 * lamda01 * lamda01 +
            quad3_0202 * lamda02 * lamda02 +
            quad3_0303 * lamda03 * lamda03 +
            quad3_0102 * lamda01 * lamda02 +
            quad3_0103 * lamda01 * lamda03 +
            quad3_0203 * lamda02 * lamda03;

    b1 = bond1 * bond1 - s01sq - quad1;
    b2 = bond2 * bond2 - s02sq - quad2;
    b3 = bond3 * bond3 - s03sq - quad3;

    lamda01_new = a11inv * b1 + a12inv * b2 + a13inv * b3;
    lamda02_new = a21inv * b1 + a22inv * b2 + a23inv * b3;
    lamda03_new = a31inv * b1 + a32inv * b2 + a33inv * b3;

    done++;
    done = (fabs(lamda01_new - lamda01) > _tolerance) ? 0 : done;
    done = (fabs(lamda02_new - lamda02) > _tolerance) ? 0 : done;
    done = (fabs(lamda03_new - lamda03) > _tolerance) ? 0 : done;

    lamda01 = done < 2 ? lamda01_new : lamda01;
    lamda02 = done < 2 ? lamda02_new : lamda02;
    lamda03 = done < 2 ? lamda03_new : lamda03;
    niter++;
  }

  // update forces if atom is owned by this processor

  lamda01 *= X_F(1.0) / _dtfsq;
  lamda02 *= X_F(1.0) / _dtfsq;
  lamda03 *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0] 			+= lamda01 * r01.x + lamda02 * r02.x + lamda03 * r03.x;
    _f[i0 + _nmax] 	+= lamda01 * r01.y + lamda02 * r02.y + lamda03 * r03.y;
    _f[i0 + 2 * _nmax] 	+= lamda01 * r01.z + lamda02 * r02.z + lamda03 * r03.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1] -= lamda01 * r01.x;
    _f[i1 + _nmax] -= lamda01 * r01.y;
    _f[i1 + 2 * _nmax] -= lamda01 * r01.z;
    list[nlist++] = i1;
  }

  if(i2 < _nlocal) {
    _f[i2] -= lamda02 * r02.x;
    _f[i2 + _nmax] -= lamda02 * r02.y;
    _f[i2 + 2 * _nmax] -= lamda02 * r02.z;
    list[nlist++] = i2;
  }

  if(i3 < _nlocal) {
    _f[i3] -= lamda03 * r03.x;
    _f[i3 + _nmax] -= lamda03 * r03.y;
    _f[i3 + 2 * _nmax] -= lamda03 * r03.z;
    list[nlist++] = i3;
  }

  if(vflag || vflag_atom) {
    ENERGY_FLOAT* shared = &sharedmem[threadIdx.x];
    X_FLOAT factor = X_F(2.0) / X_F(4.0) * nlist;
    v[0] = lamda01 * r01.x * r01.x + lamda02 * r02.x * r02.x + lamda03 * r03.x * r03.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda01 * r01.y * r01.y + lamda02 * r02.y * r02.y + lamda03 * r03.y * r03.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda01 * r01.z * r01.z + lamda02 * r02.z * r02.z + lamda03 * r03.z * r03.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda01 * r01.x * r01.y + lamda02 * r02.x * r02.y + lamda03 * r03.x * r03.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda01 * r01.x * r01.z + lamda02 * r02.x * r02.z + lamda03 * r03.x * r03.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda01 * r01.y * r01.z + lamda02 * r02.y * r02.z + lamda03 * r03.y * r03.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 4.0, v);
  }
}

__device__ void FixShakeCuda_Shake3Angle(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[3];
  ENERGY_FLOAT v[6];
  X_FLOAT invmass0, invmass1, invmass2;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  int i2 = _map_array[_shake_atom[m + 2 * _nmax]];
  X_FLOAT bond1 = _bond_distance[_shake_type[m]];
  X_FLOAT bond2 = _bond_distance[_shake_type[m + _nmax]];
  X_FLOAT bond12 = _angle_distance[_shake_type[m + 2 * _nmax]];

  // r01 = distance vec between atoms, with PBC

  X_FLOAT3 r01, r02, r12;

  X_FLOAT4 x_i0, x_i1, x_i2;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);
  x_i2 = fetchXType(i2);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  r02.x = x_i0.x - x_i2.x;
  r02.y = x_i0.y - x_i2.y;
  r02.z = x_i0.z - x_i2.z;
  minimum_image(r02);

  r12.x = x_i1.x - x_i2.x;
  r12.y = x_i1.y - x_i2.y;
  r12.z = x_i1.z - x_i2.z;
  minimum_image(r12);

  // s01 = distance vec after unconstrained update, with PBC

  X_FLOAT3 s01, s02, s12;
  X_FLOAT3 xs_i0 = _xshake[i0];
  X_FLOAT3 xs_i1 = _xshake[i1];
  X_FLOAT3 xs_i2 = _xshake[i2];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  s02.x = xs_i0.x - xs_i2.x;
  s02.y = xs_i0.y - xs_i2.y;
  s02.z = xs_i0.z - xs_i2.z;
  minimum_image(s02);

  s12.x = xs_i1.x - xs_i2.x;
  s12.y = xs_i1.y - xs_i2.y;
  s12.z = xs_i1.z - xs_i2.z;
  minimum_image(s12);

  // scalar distances between atoms

  X_FLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_FLOAT r02sq = r02.x * r02.x + r02.y * r02.y + r02.z * r02.z;
  X_FLOAT r12sq = r12.x * r12.x + r12.y * r12.y + r12.z * r12.z;
  X_FLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;
  X_FLOAT s02sq = s02.x * s02.x + s02.y * s02.y + s02.z * s02.z;
  X_FLOAT s12sq = s12.x * s12.x + s12.y * s12.y + s12.z * s12.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
    invmass2 = X_F(1.0) / _rmass[i2];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
    invmass2 = X_F(1.0) / _mass[static_cast <int>(x_i2.w)];
  }

  X_FLOAT a11 = X_F(2.0) * (invmass0 + invmass1) *
                (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_FLOAT a12 = X_F(2.0) * invmass0 *
                (s01.x * r02.x + s01.y * r02.y + s01.z * r02.z);
  X_FLOAT a13 = - X_F(2.0) * invmass1 *
                (s01.x * r12.x + s01.y * r12.y + s01.z * r12.z);
  X_FLOAT a21 = X_F(2.0) * invmass0 *
                (s02.x * r01.x + s02.y * r01.y + s02.z * r01.z);
  X_FLOAT a22 = X_F(2.0) * (invmass0 + invmass2) *
                (s02.x * r02.x + s02.y * r02.y + s02.z * r02.z);
  X_FLOAT a23 = X_F(2.0) * invmass2 *
                (s02.x * r12.x + s02.y * r12.y + s02.z * r12.z);
  X_FLOAT a31 = - X_F(2.0) * invmass1 *
                (s12.x * r01.x + s12.y * r01.y + s12.z * r01.z);
  X_FLOAT a32 = X_F(2.0) * invmass2 *
                (s12.x * r02.x + s12.y * r02.y + s12.z * r02.z);
  X_FLOAT a33 = X_F(2.0) * (invmass1 + invmass2) *
                (s12.x * r12.x + s12.y * r12.y + s12.z * r12.z);

  // inverse of matrix

  X_FLOAT determ = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 -
                   a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

  if(determ == X_F(0.0)) _flag[0]++;

  X_FLOAT determinv = X_F(1.0) / determ;

  X_FLOAT a11inv = determinv * (a22 * a33 - a23 * a32);
  X_FLOAT a12inv = -determinv * (a12 * a33 - a13 * a32);
  X_FLOAT a13inv = determinv * (a12 * a23 - a13 * a22);
  X_FLOAT a21inv = -determinv * (a21 * a33 - a23 * a31);
  X_FLOAT a22inv = determinv * (a11 * a33 - a13 * a31);
  X_FLOAT a23inv = -determinv * (a11 * a23 - a13 * a21);
  X_FLOAT a31inv = determinv * (a21 * a32 - a22 * a31);
  X_FLOAT a32inv = -determinv * (a11 * a32 - a12 * a31);
  X_FLOAT a33inv = determinv * (a11 * a22 - a12 * a21);

  // quadratic correction coeffs

  X_FLOAT r0102 = (r01.x * r02.x + r01.y * r02.y + r01.z * r02.z);
  X_FLOAT r0112 = (r01.x * r12.x + r01.y * r12.y + r01.z * r12.z);
  X_FLOAT r0212 = (r02.x * r12.x + r02.y * r12.y + r02.z * r12.z);

  X_FLOAT quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_FLOAT quad1_0202 = invmass0 * invmass0 * r02sq;
  X_FLOAT quad1_1212 = invmass1 * invmass1 * r12sq;
  X_FLOAT quad1_0102 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0102;
  X_FLOAT quad1_0112 = - X_F(2.0) * (invmass0 + invmass1) * invmass1 * r0112;
  X_FLOAT quad1_0212 = - X_F(2.0) * invmass0 * invmass1 * r0212;

  X_FLOAT quad2_0101 = invmass0 * invmass0 * r01sq;
  X_FLOAT quad2_0202 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
  X_FLOAT quad2_1212 = invmass2 * invmass2 * r12sq;
  X_FLOAT quad2_0102 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0102;
  X_FLOAT quad2_0112 = X_F(2.0) * invmass0 * invmass2 * r0112;
  X_FLOAT quad2_0212 = X_F(2.0) * (invmass0 + invmass2) * invmass2 * r0212;

  X_FLOAT quad3_0101 = invmass1 * invmass1 * r01sq;
  X_FLOAT quad3_0202 = invmass2 * invmass2 * r02sq;
  X_FLOAT quad3_1212 = (invmass1 + invmass2) * (invmass1 + invmass2) * r12sq;
  X_FLOAT quad3_0102 = - X_F(2.0) * invmass1 * invmass2 * r0102;
  X_FLOAT quad3_0112 = - X_F(2.0) * (invmass1 + invmass2) * invmass1 * r0112;
  X_FLOAT quad3_0212 = X_F(2.0) * (invmass1 + invmass2) * invmass2 * r0212;
  // iterate until converged

  X_FLOAT lamda01 = X_F(0.0);
  X_FLOAT lamda02 = X_F(0.0);
  X_FLOAT lamda12 = X_F(0.0);
  int niter = 0;
  int done = 0;

  X_FLOAT quad1, quad2, quad3, b1, b2, b3, lamda01_new, lamda02_new, lamda12_new;

  //maybe all running full loop?
  while(__any(!done) && niter < _max_iter) {
    quad1 = quad1_0101 * lamda01 * lamda01 +
            quad1_0202 * lamda02 * lamda02 +
            quad1_1212 * lamda12 * lamda12 +
            quad1_0102 * lamda01 * lamda02 +
            quad1_0112 * lamda01 * lamda12 +
            quad1_0212 * lamda02 * lamda12;

    quad2 = quad2_0101 * lamda01 * lamda01 +
            quad2_0202 * lamda02 * lamda02 +
            quad2_1212 * lamda12 * lamda12 +
            quad2_0102 * lamda01 * lamda02 +
            quad2_0112 * lamda01 * lamda12 +
            quad2_0212 * lamda02 * lamda12;

    quad3 = quad3_0101 * lamda01 * lamda01 +
            quad3_0202 * lamda02 * lamda02 +
            quad3_1212 * lamda12 * lamda12 +
            quad3_0102 * lamda01 * lamda02 +
            quad3_0112 * lamda01 * lamda12 +
            quad3_0212 * lamda02 * lamda12;

    b1 = bond1 * bond1 - s01sq - quad1;
    b2 = bond2 * bond2 - s02sq - quad2;
    b3 = bond12 * bond12 - s12sq - quad3;

    lamda01_new = a11inv * b1 + a12inv * b2 + a13inv * b3;
    lamda02_new = a21inv * b1 + a22inv * b2 + a23inv * b3;
    lamda12_new = a31inv * b1 + a32inv * b2 + a33inv * b3;

    done++;
    done = (fabs(lamda01_new - lamda01) > _tolerance) ? 0 : done;
    done = (fabs(lamda02_new - lamda02) > _tolerance) ? 0 : done;
    done = (fabs(lamda12_new - lamda12) > _tolerance) ? 0 : done;

    lamda01 = done < 2 ? lamda01_new : lamda01;
    lamda02 = done < 2 ? lamda02_new : lamda02;
    lamda12 = done < 2 ? lamda12_new : lamda12;
    niter++;
  }

  // update forces if atom is owned by this processor

  lamda01 *= X_F(1.0) / _dtfsq;
  lamda02 *= X_F(1.0) / _dtfsq;
  lamda12 *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0] 			+= lamda01 * r01.x + lamda02 * r02.x;
    _f[i0 + _nmax] 	+= lamda01 * r01.y + lamda02 * r02.y;
    _f[i0 + 2 * _nmax] 	+= lamda01 * r01.z + lamda02 * r02.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1] 			-= lamda01 * r01.x - lamda12 * r12.x;
    _f[i1 + _nmax] 	-= lamda01 * r01.y - lamda12 * r12.y;
    _f[i1 + 2 * _nmax] 	-= lamda01 * r01.z - lamda12 * r12.z;
    list[nlist++] = i1;
  }

  if(i2 < _nlocal) {
    _f[i2] 			-= lamda02 * r02.x + lamda12 * r12.x;
    _f[i2 + _nmax] 	-= lamda02 * r02.y + lamda12 * r12.y;
    _f[i2 + 2 * _nmax] 	-= lamda02 * r02.z + lamda12 * r12.z;
    list[nlist++] = i2;
  }

  if(vflag || vflag_atom) {
    ENERGY_FLOAT* shared = &sharedmem[threadIdx.x];
    X_FLOAT factor = X_F(2.0) / X_F(3.0) * nlist;
    v[0] = lamda01 * r01.x * r01.x + lamda02 * r02.x * r02.x + lamda12 * r12.x * r12.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda01 * r01.y * r01.y + lamda02 * r02.y * r02.y + lamda12 * r12.y * r12.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda01 * r01.z * r01.z + lamda02 * r02.z * r02.z + lamda12 * r12.z * r12.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda01 * r01.x * r01.y + lamda02 * r02.x * r02.y + lamda12 * r12.x * r12.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda01 * r01.x * r01.z + lamda02 * r02.x * r02.z + lamda12 * r12.x * r12.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda01 * r01.y * r01.z + lamda02 * r02.y * r02.z + lamda12 * r12.y * r12.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 3.0, v);
  }
}

__global__ void FixShakeCuda_Shake_Kernel(int vflag, int vflag_atom, int* list, int nlist)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < nlist) {

    int m = list[i];
    int sflag = _shake_flag[m];

    if(sflag == 2) FixShakeCuda_Shake2(vflag, vflag_atom, m);
    else if(sflag == 3) FixShakeCuda_Shake3(vflag, vflag_atom, m);
    else if(sflag == 4) FixShakeCuda_Shake4(vflag, vflag_atom, m);
    else FixShakeCuda_Shake3Angle(vflag, vflag_atom, m);
  } else {
    ENERGY_FLOAT* shared = &sharedmem[threadIdx.x];
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
  }

  if(vflag) {
    __syncthreads();
    int eflag = 0;
    PairVirialCompute_A_Kernel(eflag, vflag);
  }

}

__global__ void FixShakeCuda_PackComm_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_FLOAT dx, X_FLOAT dy, X_FLOAT dz)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    X_FLOAT3 xs = _xshake[j];
    ((X_FLOAT*) _buffer)[i] = xs.x + dx;
    ((X_FLOAT*) _buffer)[i + 1 * n] = xs.y + dy;
    ((X_FLOAT*) _buffer)[i + 2 * n] = xs.z + dz;
  }

}

__global__ void FixShakeCuda_PackComm_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_FLOAT dx, X_FLOAT dy, X_FLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    X_FLOAT3 xs = _xshake[j];
    xs.x += dx;
    xs.y += dy;
    xs.z += dz;
    _xshake[i + first] = xs;
  }

}

__global__ void FixShakeCuda_UnpackComm_Kernel(int n, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    X_FLOAT3 xs;
    xs.x = ((X_FLOAT*) _buffer)[i];
    xs.y = ((X_FLOAT*) _buffer)[i + 1 * n];
    xs.z = ((X_FLOAT*) _buffer)[i + 2 * n];
    _xshake[i + first] = xs;
  }
}

